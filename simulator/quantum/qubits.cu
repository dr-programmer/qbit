#include "hip/hip_runtime.h"
#include "qubits_cuda.h"
extern "C" {
#define SMART_DEALLOCATION
#include "smart_allocation.h"
}

#include <stdio.h>
#include <stdlib.h>

__global__ void quantum_state_measure_kernel(const struct complex * const state, 
                                                unsigned int * const result,  
                                                const size_t length, 
                                                const float random_value, 
                                                float * const current_state_probability, 
                                                int * const done) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length || atomicAdd(done, 0) == 1) return;

    float probability = state[index].real * state[index].real 
                        + state[index].imaginary * state[index].imaginary;
    float accumulated_probability = atomicAdd(current_state_probability, probability);

    if(accumulated_probability + probability > random_value) {
        if(atomicExch(done, 1) == 0) {
            atomicExch(result, index);
        }
    }
}

extern "C" qm_result *quantum_state_measure_cuda(const quantum_state * const q) {
    if(!q) {
        fprintf(stderr, "Error: quantum_state_measure_cuda: quantum_state is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring state:\n");
    matrix_print(q);
    #endif

    size_t length = q->rows;
    size_t size = sizeof(struct complex) * length;

    struct complex *cuda_q;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_q, size));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_q, q->fields, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("To kernel with random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    quantum_state_measure_kernel<<<blocksPerGrid, DTCPB>>>(cuda_q, cuda_result, 
                        length, random_value, cuda_current_state_probability, cuda_done);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    quantum_state *result_state = quantum_state_create(result, length);
    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_q));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}

__global__ void quantum_state_measure_density_matrix_kernel(
                                                const struct complex * const state, 
                                                unsigned int * const result,  
                                                const size_t length, 
                                                const float random_value, 
                                                float * const current_state_probability, 
                                                int * const done) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length || atomicAdd(done, 0) == 1) return;

    unsigned int field_index = (index * length) + index;
    float probability = state[field_index].real;
    float accumulated_probability = atomicAdd(current_state_probability, probability);

    if(accumulated_probability + probability > random_value) {
        if(atomicExch(done, 1) == 0) {
            atomicExch(result, index);
        }
    }
}

extern "C" qm_result *quantum_state_measure_density_matrix_cuda(
                                                const quantum_operator * const d) 
{
    if(!d) {
        fprintf(stderr, 
            "Error: quantum_state_measure_density_matrix_cuda: quantum_operator is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_density_matrix_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring density matrix:\n");
    matrix_print(d);
    #endif

    size_t length_m1 = d->rows * d->columns;
    size_t size_m1 = sizeof(struct complex) * length_m1;
    size_t length = d->rows;

    struct complex *cuda_d;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_d, size_m1));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_d, d->fields, size_m1, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("To kernel with random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    quantum_state_measure_density_matrix_kernel<<<blocksPerGrid, DTCPB>>>(cuda_d, 
                                                cuda_result, length, random_value, 
                                                cuda_current_state_probability, cuda_done);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    quantum_state *result_state = quantum_state_create(result, length);
    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_d));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}

__global__ void quantum_state_measure_subsystem_kernel(const struct complex * const state, 
                                            unsigned int * const result, 
                                            const size_t length, 
                                            const float random_value, 
                                            float * const current_state_probability, 
                                            int * const done, 
                                            const unsigned int measured_qubits_count, 
                                            const unsigned int preceding_qubits_count,  
                                            const unsigned int proceeding_qubits_count) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length || atomicAdd(done, 0) == 1) return;

    /* 
    In a quantum system, the state of n qubits is represented as an index in binary, 
    where each bit corresponds to a qubit's state (0 or 1). For example, in a 3-qubit 
    system, the index '101' (binary) corresponds to the state |101>. To measure a 
    subset of qubits, we shift the index right by the number of proceeding qubits so 
    that the measured qubits are aligned to the least significant bits (the rightmost bits). 
    Then, a bitwise AND operation is applied with a mask that isolates the bits corresponding 
    to the measured qubits. The mask is generated by (1 << measured_qubits_count) - 1, 
    which creates a binary number with the least significant bits set to 1, matching 
    the number of measured qubits. This operation effectively extracts the group ID 
    corresponding to the measured qubits, ignoring the bits for the preceding and 
    proceeding qubits. 
    */
    unsigned int group_id = (index >> proceeding_qubits_count) 
                                & ((1 << measured_qubits_count) - 1);

    float magnitude_squared = state[index].real * state[index].real 
        + state[index].imaginary * state[index].imaginary;

    atomicAdd(&current_state_probability[group_id], magnitude_squared);

    __syncthreads(); // Ensure all threads finish probability computation

    #ifdef QUBITS_CUDA_SHOW_KERNEL_OPERATIONS
    printf("Index: %u\n", index);
    printf("Group ID: %u for index %u\n", group_id, index);
    printf("Magnitude squared: %f for index %u\n", magnitude_squared, index);
    printf("Probability: %f for group id %u\n", 
        current_state_probability[group_id], group_id);
    #endif

    if(index == 0) {
        float cumulative_probability = 0;
        const unsigned int possible_outputs = 1 << measured_qubits_count;
        for(unsigned int i = 0; i < possible_outputs; ++i) {
            cumulative_probability += current_state_probability[i];
            if(random_value < cumulative_probability) {
                atomicExch(result, i);
                atomicExch(done, 1);
                break;
            }
        }
    }
}

__global__ void quantum_state_collapse_to_subsystem_kernel(struct complex * const state, 
                                            const unsigned int result, 
                                            const size_t length, 
                                            const float * const current_state_probability, 
                                            const unsigned int measured_qubits_count, 
                                            const unsigned int preceding_qubits_count,  
                                            const unsigned int proceeding_qubits_count) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length) return;

    // The explanation for the group ID calculation is the same as in the measurement kernel
    unsigned int group_id = (index >> proceeding_qubits_count) 
                                & ((1 << measured_qubits_count) - 1);

    // Zero out amplitudes not belonging to the measured group
    if(group_id != result) {
        state[index].real = 0;
        state[index].imaginary = 0;
    }
    else {
        // Normalize the remaining amplitudes
        float norm_factor = sqrtf(current_state_probability[result]);
        if(norm_factor > 0) {
            state[index].real /= norm_factor;
            state[index].imaginary /= norm_factor;
        }
    }
}

extern "C" qm_result *quantum_state_measure_subsystem_cuda(const quantum_state * const q, 
                                                const unsigned int start_system_index, 
                                                const unsigned int end_system_index) 
{
    if(!q) {
        fprintf(stderr, 
            "Error: quantum_state_measure_subsystem_cuda: quantum_state is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_subsystem_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring subsystem:\n");
    matrix_print(q);
    #endif

    const unsigned int systems = log2(q->rows);

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Systems: %u \n", systems);
    #endif

    const unsigned int preceding_qubits_count = start_system_index;
    const unsigned int measured_qubits_count = 
        (end_system_index - start_system_index) + 1;
    const unsigned int proceeding_qubits_count = 
        systems - (end_system_index + 1);

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Measured qubits: %u \n", measured_qubits_count);
    #endif

    const unsigned int possible_outputs = 1 << measured_qubits_count;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Possible outputs: %u \n", possible_outputs);
    #endif

    size_t length_q = q->rows;
    size_t size_q = sizeof(struct complex) * length_q;

    struct complex *cuda_q;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_q, size_q));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, possible_outputs * sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_q, q->fields, size_q, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, 
                            possible_outputs * sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length_q + DTCPB - 1) / DTCPB;

    quantum_state_measure_subsystem_kernel<<<blocksPerGrid, DTCPB>>>(cuda_q, cuda_result, 
                        length_q, random_value, cuda_current_state_probability, 
                        cuda_done, measured_qubits_count, preceding_qubits_count, 
                        proceeding_qubits_count);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Measured result: %u\n", result);
    #endif

    quantum_state_collapse_to_subsystem_kernel<<<blocksPerGrid, DTCPB>>>(cuda_q, 
                        result, length_q, cuda_current_state_probability, 
                        measured_qubits_count, preceding_qubits_count, 
                        proceeding_qubits_count);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    quantum_state *result_state = quantum_state_create(0, length_q);
    CUDA_CHECK(hipMemcpy(result_state->fields, cuda_q, size_q, hipMemcpyDeviceToHost));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Collapsed state:\n");
    matrix_print(result_state);
    #endif

    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_q));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}
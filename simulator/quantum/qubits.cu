#include "hip/hip_runtime.h"
#include "qubits_cuda.h"
extern "C" {
#define SMART_DEALLOCATION
#include "smart_allocation.h"
}

#include <stdio.h>
#include <stdlib.h>

__global__ void quantum_state_measure_kernel(const struct complex * const state, 
                                                unsigned int * const result,  
                                                const size_t length, 
                                                const float random_value, 
                                                float * const current_state_probability, 
                                                int * const done) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length || atomicAdd(done, 0) == 1) return;

    float probability = state[index].real * state[index].real 
                        + state[index].imaginary * state[index].imaginary;
    float accumulated_probability = atomicAdd(current_state_probability, probability);

    if(accumulated_probability + probability >= random_value) {
        if(atomicExch(done, 1) == 0) {
            atomicExch(result, index);
        }
    }
}

extern "C" qm_result *quantum_state_measure_cuda(const quantum_state * const q) {
    if(!q) {
        fprintf(stderr, "Error: quantum_state_measure_cuda: quantum_state is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring state:\n");
    matrix_print(q);
    #endif

    size_t length = q->rows;
    size_t size = sizeof(struct complex) * length;

    struct complex *cuda_q;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_q, size));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_q, q->fields, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("To kernel with random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    quantum_state_measure_kernel<<<blocksPerGrid, DTCPB>>>(cuda_q, cuda_result, 
                        length, random_value, cuda_current_state_probability, cuda_done);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    quantum_state *result_state = quantum_state_create(result, length);
    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_q));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}

__global__ void quantum_state_measure_density_matrix_kernel(
                                                const struct complex * const state, 
                                                unsigned int * const result,  
                                                const size_t length, 
                                                const float random_value, 
                                                float * const current_state_probability, 
                                                int * const done) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length || atomicAdd(done, 0) == 1) return;

    unsigned int field_index = (index * length) + index;
    float probability = state[field_index].real;
    float accumulated_probability = atomicAdd(current_state_probability, probability);

    if(accumulated_probability + probability >= random_value) {
        if(atomicExch(done, 1) == 0) {
            atomicExch(result, index);
        }
    }
}

extern "C" qm_result *quantum_state_measure_density_matrix_cuda(
                                                const quantum_operator * const d) 
{
    if(!d) {
        fprintf(stderr, 
            "Error: quantum_state_measure_density_matrix_cuda: quantum_operator is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_density_matrix_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring density matrix:\n");
    matrix_print(d);
    #endif

    size_t length_m1 = d->rows * d->columns;
    size_t size_m1 = sizeof(struct complex) * length_m1;
    size_t length = d->rows;

    struct complex *cuda_d;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_d, size_m1));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_d, d->fields, size_m1, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("To kernel with random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    quantum_state_measure_density_matrix_kernel<<<blocksPerGrid, DTCPB>>>(cuda_d, 
                                                cuda_result, length, random_value, 
                                                cuda_current_state_probability, cuda_done);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    quantum_state *result_state = quantum_state_create(result, length);
    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_d));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}

__global__ void quantum_state_measure_subsystem_kernel(...) 
{
    // To be implemented
}

extern "C" qm_result *quantum_state_measure_subsystem_cuda(const quantum_state * const q, 
                                                const unsigned int start_system_index, 
                                                const unsigned int end_system_index) 
{
    if(!q) {
        fprintf(stderr, 
            "Error: quantum_state_measure_subsystem_cuda: quantum_state is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("quantum_state_measure_subsystem_cuda\n");
    #endif

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Measuring subsystem:\n");
    matrix_print(q);
    #endif

    quantum_operator *density_matrix = NULL;
    if(q->rows == q->columns) density_matrix = (quantum_operator *)q;
    else density_matrix = matrix_mul(q, vector_get_dual(q));

    if(!density_matrix) {
        fprintf(stderr, 
            "Error: quantum_state_measure_subsystem_cuda: density matrix is NULL\n");
        return NULL;
    }

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS_THOROUGH
    printf("Density matrix:\n");
    matrix_print(d);
    #endif

    const unsigned int systems = log2(q->rows);

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Systems: %u \n", systems);
    #endif

    const unsigned int preceding_qubits_count = start_system_index;
    const unsigned int measured_qubits_count = 
        (end_system_index - start_system_index) + 1;
    const unsigned int proceeding_qubits_count = 
        systems - (end_system_index + 1);

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Measured qubits: %u \n", measured_qubits_count);
    #endif

    const unsigned int possible_outputs = pow(2, measured_qubits_count);

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Possible outputs: %u \n", possible_outputs);
    #endif

    size_t length_d = density_matrix->rows * density_matrix->columns;
    size_t size_d = sizeof(struct complex) * length_d;

    struct complex *cuda_d;
    unsigned int *cuda_result;
    float *cuda_current_state_probability;
    int *cuda_done;
    CUDA_CHECK(hipMalloc(&cuda_d, size_d));
    CUDA_CHECK(hipMalloc(&cuda_result, sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc(&cuda_current_state_probability, sizeof(float)));
    CUDA_CHECK(hipMalloc(&cuda_done, sizeof(int)));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_d, density_matrix->fields, size_d, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, sizeof(unsigned int)));
    CUDA_CHECK(hipMemset(cuda_current_state_probability, 0, sizeof(float)));
    CUDA_CHECK(hipMemset(cuda_done, 0, sizeof(int)));

    float random_value = (float)rand() / RAND_MAX;

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Random value: %f\n", random_value);
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (possible_outputs + DTCPB - 1) / DTCPB;
    quantum_state_measure_subsystem_kernel<<<blocksPerGrid, DTCPB>>>();
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("After kernel\n");
    #endif

    unsigned int result;
    CUDA_CHECK(hipMemcpy(&result, cuda_result, 
                            sizeof(unsigned int), hipMemcpyDeviceToHost));

    quantum_state *result_state = quantum_state_create(result, possible_outputs);

    qm_result *qm_result = qm_result_create(result_state, result);

    CUDA_CHECK(hipFree(cuda_d));
    CUDA_CHECK(hipFree(cuda_result));
    CUDA_CHECK(hipFree(cuda_current_state_probability));
    CUDA_CHECK(hipFree(cuda_done));

    #ifdef QUBITS_CUDA_SHOW_OPERATIONS
    printf("Freed\n");
    #endif

    return qm_result;
}
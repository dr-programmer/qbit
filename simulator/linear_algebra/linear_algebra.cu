#include "hip/hip_runtime.h"
#include "linear_algebra_cuda.h"
extern "C" {
#define SMART_DEALLOCATION
#include "smart_allocation.h"
}

#include <stdio.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", \
                                __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(err); \
        } \
    }

__global__ void matrix_add_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length) return;

    result[index] = {m1[index].real + m2[index].real, 
                        m1[index].imaginary + m2[index].imaginary};
}
__global__ void matrix_sub_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= length) return;

    result[index] = {m1[index].real - m2[index].real, 
                        m1[index].imaginary - m2[index].imaginary};
}
__global__ void matrix_mul_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t rows, size_t columns, size_t shared_dim) 
{
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int column = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= rows || column >= columns) return;

    struct complex sum = {0, 0};
    for(size_t i = 0; i < shared_dim; i++) {
        struct complex a = m1[row * shared_dim + i];
        struct complex b = m2[i * columns + column];
        sum.real += a.real * b.real - a.imaginary * b.imaginary;
        sum.imaginary += a.real * b.imaginary + a.imaginary * b.real;
    }

    result[row * columns + column] = sum;
}
__global__ void matrix_mul_scalar_kernel(const struct complex s, 
                                            const struct complex * const m, 
                                            struct complex * const result, 
                                            size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length) return;

    result[index] = {s.real * m[index].real - s.imaginary * m[index].imaginary, 
                        s.real * m[index].imaginary + s.imaginary * m[index].real};
}

__global__ void print_array(struct complex *array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("array[%d] = %f + %fi\n", idx, array[idx].real, array[idx].imaginary);
    }
}
struct matrix *matrix_operation_cuda(const void *m1_s_void, 
                                        const void *m2_void, 
                                        const int operation) 
{
    if(!m1_s_void || !m2_void) return NULL;

    const struct matrix *m1, *m2;
    struct complex s;

    if(operation == OPERATION_MUL_SCALAR) s = *(const struct complex *)m1_s_void;
    else m1 = (const struct matrix *)m1_s_void;
    m2 = (const struct matrix *)m2_void;

    switch(operation) {
        case OPERATION_ADD ... OPERATION_SUB:
            if(m1->rows != m2->rows || m1->columns != m2->columns) return NULL;
            break;
        case OPERATION_MUL:
            if(m1->columns != m2->rows) return NULL;
            break;
        case OPERATION_MUL_SCALAR:
            break;
        default:
            return NULL;
    }

    // length = the number of complex numbers in the matrix
    // size = the size of the matrix in bytes
    size_t length_m1 = 0;
    size_t size_m1 = 0;
    size_t length_m2 = m2->rows * m2->columns;
    size_t size_m2 = sizeof(struct complex) * length_m2;
    size_t length;
    size_t size;

    if(operation == OPERATION_MUL_SCALAR) {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Mul scalar\n");
        #endif
        length = length_m2;
        size = size_m2;
    }
    else if(operation == OPERATION_MUL) {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Mul\n");
        #endif
        length_m1 = m1->rows * m1->columns;
        size_m1 = sizeof(struct complex) * length_m1;

        length = m1->rows * m2->columns;
        size = sizeof(struct complex) * length;
    }
    else {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Add or sub\n");
        #endif
        length_m1 = m1->rows * m1->columns;
        size_m1 = sizeof(struct complex) * length_m1;

        length = length_m1;
        size = size_m1;
    }

    struct complex *cuda_m1, *cuda_m2, *cuda_result;
    if(operation != OPERATION_MUL_SCALAR) CUDA_CHECK(hipMalloc(&cuda_m1, size_m1));
    CUDA_CHECK(hipMalloc(&cuda_m2, size_m2));
    CUDA_CHECK(hipMalloc(&cuda_result, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Allocated\n");
    #endif

    if(operation != OPERATION_MUL_SCALAR) 
        CUDA_CHECK(hipMemcpy(cuda_m1, m1->fields, size_m1, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_m2, m2->fields, size_m2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("To kernel\n");
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    dim3 threadsPerBlock2D(DTCPB2D, DTCPB2D);
    dim3 blocksPerGrid2D;
    if(operation == OPERATION_MUL) 
        blocksPerGrid2D = dim3((m2->columns + threadsPerBlock2D.x - 1) / threadsPerBlock2D.x, 
                                (m1->rows + threadsPerBlock2D.y - 1) / threadsPerBlock2D.y);
    if(operation == OPERATION_ADD) 
        matrix_add_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, cuda_result, length);
    else if(operation == OPERATION_SUB) 
        matrix_sub_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, cuda_result, length);
    else if(operation == OPERATION_MUL) 
        matrix_mul_kernel<<<blocksPerGrid2D, threadsPerBlock2D>>>(cuda_m1, cuda_m2, 
                                                        cuda_result, 
                                                        m1->rows, m2->columns, m1->columns);
    else if(operation == OPERATION_MUL_SCALAR) 
        matrix_mul_scalar_kernel<<<blocksPerGrid, DTCPB>>>(s, cuda_m2, cuda_result, length);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("After kernel\n");
    #endif

    struct matrix *result;
    if(operation == OPERATION_MUL) result = matrix_create_empty(m1->rows, m2->columns);
    else result = matrix_create_empty(m2->rows, m2->columns);
    CUDA_CHECK(hipMemcpy(result->fields, cuda_result, size, hipMemcpyDeviceToHost));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS_THOROUGH
    printf("result:\n");
    matrix_print(result);
    printf("For:\n");
    operation != OPERATION_MUL_SCALAR ? matrix_print(m1) : complex_print(s);
    if(operation == OPERATION_MUL_SCALAR) printf("\n");
    printf("And:\n");
    matrix_print(m2);
    printf("M1\n");
    if(operation != OPERATION_MUL_SCALAR) 
        print_array<<<blocksPerGrid, DTCPB>>>(cuda_m1, length_m1);
    else {
        complex_print(s);
        printf("\n");
    }
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M2\n");
    print_array<<<blocksPerGrid, DTCPB>>>(cuda_m2, length_m2);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("Result\n");
    print_array<<<blocksPerGrid, DTCPB>>>(cuda_result, length);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M1 length: %zu\n", length_m1);
    printf("M2 length: %zu\n", length_m2);
    #endif

    if(operation != OPERATION_MUL_SCALAR) hipFree(cuda_m1);
    hipFree(cuda_m2);
    hipFree(cuda_result);

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Freed\n");
    #endif

    return result;
}
extern "C" struct matrix *matrix_add_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_ADD);
}
extern "C" struct matrix *matrix_sub_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_SUB);
}
extern "C" struct matrix *matrix_mul_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_MUL);
}
extern "C" struct matrix *matrix_mul_scalar_cuda(const struct complex s, 
                                    const struct matrix * const m) 
{
    return matrix_operation_cuda(&s, m, OPERATION_MUL_SCALAR);
}
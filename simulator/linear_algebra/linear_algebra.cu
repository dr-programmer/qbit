#include "hip/hip_runtime.h"
#include "linear_algebra_cuda.h"
extern "C" {
#define SMART_DEALLOCATION
#include "smart_allocation.h"
}

#include <stdio.h>

__global__ void matrix_add_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length) return;

    result[index] = {m1[index].real + m2[index].real, 
                        m1[index].imaginary + m2[index].imaginary};
}
__global__ void matrix_sub_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= length) return;

    result[index] = {m1[index].real - m2[index].real, 
                        m1[index].imaginary - m2[index].imaginary};
}
__global__ void matrix_mul_kernel(const struct complex * const m1, 
                                    const struct complex * const m2, 
                                    struct complex * const result, 
                                    size_t rows, size_t columns, size_t shared_dim) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= rows * columns) return;

    unsigned int row = index / columns;
    unsigned int column = index % columns;

    struct complex sum = {0, 0};
    for(size_t i = 0; i < shared_dim; i++) {
        struct complex a = m1[row * shared_dim + i];
        struct complex b = m2[i * columns + column];
        sum.real += a.real * b.real - a.imaginary * b.imaginary;
        sum.imaginary += a.real * b.imaginary + a.imaginary * b.real;
    }

    result[row * columns + column] = sum;
}
__global__ void matrix_mul_scalar_kernel(const struct complex s, 
                                            const struct complex * const m, 
                                            struct complex * const result, 
                                            size_t length) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= length) return;

    result[index] = {s.real * m[index].real - s.imaginary * m[index].imaginary, 
                        s.real * m[index].imaginary + s.imaginary * m[index].real};
}

__global__ void print_array(struct complex *array, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("array[%d] = %f + %fi\n", idx, array[idx].real, array[idx].imaginary);
    }
}
struct matrix *matrix_operation_cuda(const void *m1_s_void, 
                                        const void *m2_void, 
                                        const enum operation operation) 
{
    if(!m1_s_void || !m2_void) return NULL;

    const struct matrix *m1, *m2;
    struct complex s;

    if(operation == OPERATION_MUL_SCALAR) s = *(const struct complex *)m1_s_void;
    else m1 = (const struct matrix *)m1_s_void;
    m2 = (const struct matrix *)m2_void;

    switch(operation) {
        case OPERATION_ADD ... OPERATION_SUB:
            if(m1->rows != m2->rows || m1->columns != m2->columns) return NULL;
            break;
        case OPERATION_MUL:
            if(m1->columns != m2->rows) return NULL;
            break;
        case OPERATION_MUL_SCALAR:
            break;
        default:
            return NULL;
    }

    // length = the number of complex numbers in the matrix
    // size = the size of the matrix in bytes
    size_t length_m1 = 0;
    size_t size_m1 = 0;
    size_t length_m2 = m2->rows * m2->columns;
    size_t size_m2 = sizeof(struct complex) * length_m2;
    size_t length;
    size_t size;

    if(operation == OPERATION_MUL_SCALAR) {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Mul scalar\n");
        #endif
        length = length_m2;
        size = size_m2;
    }
    else if(operation == OPERATION_MUL) {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Mul\n");
        #endif
        length_m1 = m1->rows * m1->columns;
        size_m1 = sizeof(struct complex) * length_m1;

        length = m1->rows * m2->columns;
        size = sizeof(struct complex) * length;
    }
    else {
        #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
        printf("Add or sub\n");
        #endif
        length_m1 = m1->rows * m1->columns;
        size_m1 = sizeof(struct complex) * length_m1;

        length = length_m1;
        size = size_m1;
    }

    struct complex *cuda_m1, *cuda_m2, *cuda_result;
    if(operation != OPERATION_MUL_SCALAR) CUDA_CHECK(hipMalloc(&cuda_m1, size_m1));
    CUDA_CHECK(hipMalloc(&cuda_m2, size_m2));
    CUDA_CHECK(hipMalloc(&cuda_result, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Allocated\n");
    #endif

    if(operation != OPERATION_MUL_SCALAR) 
        CUDA_CHECK(hipMemcpy(cuda_m1, m1->fields, size_m1, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_m2, m2->fields, size_m2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("To kernel\n");
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    if(operation == OPERATION_ADD) 
        matrix_add_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, cuda_result, length);
    else if(operation == OPERATION_SUB) 
        matrix_sub_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, cuda_result, length);
    else if(operation == OPERATION_MUL) 
        matrix_mul_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, cuda_result, 
                                                        m1->rows, m2->columns, m1->columns);
    else if(operation == OPERATION_MUL_SCALAR) 
        matrix_mul_scalar_kernel<<<blocksPerGrid, DTCPB>>>(s, cuda_m2, cuda_result, length);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("After kernel\n");
    #endif

    struct matrix *result;
    if(operation == OPERATION_MUL) result = matrix_create_empty(m1->rows, m2->columns);
    else result = matrix_create_empty(m2->rows, m2->columns);
    CUDA_CHECK(hipMemcpy(result->fields, cuda_result, size, hipMemcpyDeviceToHost));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS_THOROUGH
    printf("result:\n");
    matrix_print(result);
    printf("For:\n");
    operation != OPERATION_MUL_SCALAR ? matrix_print(m1) : complex_print(s);
    if(operation == OPERATION_MUL_SCALAR) printf("\n");
    printf("And:\n");
    matrix_print(m2);
    printf("M1\n");
    if(operation != OPERATION_MUL_SCALAR) 
        print_array<<<blocksPerGrid, DTCPB>>>(cuda_m1, length_m1);
    else {
        complex_print(s);
        printf("\n");
    }
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M2\n");
    print_array<<<blocksPerGrid, DTCPB>>>(cuda_m2, length_m2);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("Result\n");
    print_array<<<blocksPerGrid, DTCPB>>>(cuda_result, length);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M1 length: %zu\n", length_m1);
    printf("M2 length: %zu\n", length_m2);
    #endif

    if(operation != OPERATION_MUL_SCALAR) CUDA_CHECK(hipFree(cuda_m1));
    CUDA_CHECK(hipFree(cuda_m2));
    CUDA_CHECK(hipFree(cuda_result));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Freed\n");
    #endif

    return result;
}
extern "C" struct matrix *matrix_add_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_ADD);
}
extern "C" struct matrix *matrix_sub_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_SUB);
}
extern "C" struct matrix *matrix_mul_cuda(const struct matrix * const m1, 
                                    const struct matrix * const m2) 
{
    return matrix_operation_cuda(m1, m2, OPERATION_MUL);
}
extern "C" struct matrix *matrix_mul_scalar_cuda(const struct complex s, 
                                    const struct matrix * const m) 
{
    return matrix_operation_cuda(&s, m, OPERATION_MUL_SCALAR);
}

__global__ void matrix_tensor_product_kernel(const struct complex * const m1, 
                                                const struct complex * const m2, 
                                                struct complex * const result, 
                                                size_t rows, size_t columns, 
                                                size_t m1_rows, size_t m1_columns, 
                                                size_t m2_rows, size_t m2_columns) 
{
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= rows * columns) return;

    unsigned int row = index / columns;
    unsigned int column = index % columns;

    struct complex a = m1[(row / m2_rows) * m1_columns + (column / m2_columns)];
    struct complex b = m2[(row % m2_rows) * m2_columns + (column % m2_columns)];

    result[row * columns + column] = {a.real * b.real - a.imaginary * b.imaginary, 
                                        a.real * b.imaginary + a.imaginary * b.real};
}

extern "C" struct matrix *matrix_tensor_product_cuda(const struct matrix * const m1, 
                                                        const struct matrix * const m2) 
{
    if(!m1 || !m2) return NULL;

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Tensor product\n");
    #endif

    size_t rows = m1->rows * m2->rows;
    size_t columns = m1->columns * m2->columns;

    size_t length_m1 = m1->rows * m1->columns;
    size_t size_m1 = sizeof(struct complex) * length_m1;
    size_t length_m2 = m2->rows * m2->columns;
    size_t size_m2 = sizeof(struct complex) * length_m2;
    size_t length = length_m1 * length_m2;
    size_t size = sizeof(struct complex) * length;

    struct complex *cuda_m1, *cuda_m2, *cuda_result;
    CUDA_CHECK(hipMalloc(&cuda_m1, size_m1));
    CUDA_CHECK(hipMalloc(&cuda_m2, size_m2));
    CUDA_CHECK(hipMalloc(&cuda_result, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Allocated\n");
    #endif

    CUDA_CHECK(hipMemcpy(cuda_m1, m1->fields, size_m1, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(cuda_m2, m2->fields, size_m2, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(cuda_result, 0, size));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("To kernel\n");
    #endif

    // Ensures the number of blocks is enough by rounding up
    int blocksPerGrid = (length + DTCPB - 1) / DTCPB;
    matrix_tensor_product_kernel<<<blocksPerGrid, DTCPB>>>(cuda_m1, cuda_m2, 
                                                                    cuda_result, 
                                                                    rows, columns, 
                                                                    m1->rows, m1->columns, 
                                                                    m2->rows, m2->columns);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("After kernel\n");
    #endif

    struct matrix *result = matrix_create_empty(rows, columns);
    CUDA_CHECK(hipMemcpy(result->fields, cuda_result, size, hipMemcpyDeviceToHost));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS_THOROUGH
    printf("result:\n");
    matrix_print(result);
    printf("For:\n");
    matrix_print(m1);
    printf("And:\n");
    matrix_print(m2);
    printf("M1\n");
    print_array<<<blocksPerGrid2D, threadsPerBlock2D>>>(cuda_m1, length_m1);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M2\n");
    print_array<<<blocksPerGrid2D, threadsPerBlock2D>>>(cuda_m2, length_m2);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("Result\n");
    print_array<<<blocksPerGrid2D, threadsPerBlock2D>>>(cuda_result, length);
    CUDA_CHECK(hipDeviceSynchronize());
    printf("M1 length: %zu\n", length_m1);
    printf("M2 length: %zu\n", length_m2);
    #endif

    CUDA_CHECK(hipFree(cuda_m1));
    CUDA_CHECK(hipFree(cuda_m2));
    CUDA_CHECK(hipFree(cuda_result));

    #ifdef LINEAR_ALGEBRA_CUDA_SHOW_OPERATION_CALCULATIONS
    printf("Freed\n");
    #endif

    return result;
}